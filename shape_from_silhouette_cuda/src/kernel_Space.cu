#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 

#include <>
#include <device_atomic_functions.h>
#include <surface_functions.h>

#include "cuda_math_utils.h"
#include "cuda_error_check.h"

#include "kernel_space.cuh"
#include "kernel_convexHull.cuh"



__device__ inline bool isValidVoxelIdx(uint x, uint y, uint z, uint offset, uint3 numVoxel)
{
	return x < numVoxel.x && y < numVoxel.y && z < numVoxel.z && offset < numVoxel.x * numVoxel.y * numVoxel.z;
}


__device__ inline bool toValidImageIdx(float x, float y, uint * retX, uint * retY, uint2 sizeImg)
{
	if(x < 0 || x >= sizeImg.x - 1 || y < 0 && y >= sizeImg.y - 1)
	{
		*retX = 0;
		*retY = 0;
		return false;
	}

	*retX = (uint)(x + 0.5f);
	*retY = (uint)(y + 0.5f);

	return true;
}


__device__ inline bool isValidImageIdx(float x, float y, uint2 sizeImg)
{
	if(x < 0 || x >= sizeImg.x || y < 0 && y >= sizeImg.y)
	{
		return false;
	}

	return true;
}


__global__ void updateSpaceFast(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxel, unsigned char * pImages, uint2 sizeImages, uint numImages, float minSegmentation)
{
	const uint voxelId_x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint voxelId_y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint voxelId_z = blockIdx.z;
	const uint camIndex  = threadIdx.z;
	
	const uint offset = voxelId_x * numVoxel.y * numVoxel.z + voxelId_y * numVoxel.z + voxelId_z; 

	if(isValidVoxelIdx(voxelId_x, voxelId_y, voxelId_z, offset, numVoxel))
	{
		VoxelSegmentationInfo & v = pVoxelInfo[offset];

		const TemporaryDeviceVector<uint2> & imgPointsSingleCam = v.getImgPoints(camIndex);
		const unsigned char * pCurrentImage = pImages + camIndex * (sizeImages.x * sizeImages.y);

		uint segmentationCounter = 0;

		assert(imgPointsSingleCam.size() % 2 == 0);
		for(size_t i = 0; i < imgPointsSingleCam.size(); i += 2)
		{
			assert(imgPointsSingleCam[i].y == imgPointsSingleCam[i+1].y);
			for(int x = imgPointsSingleCam[i].x; x <= imgPointsSingleCam[i+1].x; ++x)
			{
				const uint imgOffset = imgPointsSingleCam[i].y * sizeImages.x + x;
				const unsigned char * currentPixel = pCurrentImage + imgOffset;

				if(*currentPixel != 0)
				{
					++segmentationCounter;
				}
			}
		}
		const uint segmentationThreshold = v.getNumImgPoints(camIndex) * minSegmentation;

		const VoxelSegmentationStatus s = segmentationCounter > segmentationThreshold ? Marked : NotMarked;
		v.m_segmentationStats[camIndex] = s;
	}

	__syncthreads();

	if(isValidVoxelIdx(voxelId_x, voxelId_y, voxelId_z, offset, numVoxel) && camIndex == 0)
	{		
		VoxelSegmentationInfo & v = pVoxelInfo[offset];
		const bool active = voxelPredicate_maximumVisibleActive(v);
		pVoxel[offset].m_isActive = active;
	}
}


__global__ void updateSpaceFromIntegralImage(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxel, uint * pImages, uint2 sizeImages, float minSegmentation)
{
	const uint voxelId_x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint voxelId_y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint voxelId_z = blockIdx.z * blockDim.z + threadIdx.z;
	
	const uint offset = voxelId_x * numVoxel.y * numVoxel.z + voxelId_y * numVoxel.z + voxelId_z; 

	if(isValidVoxelIdx(voxelId_x, voxelId_y, voxelId_z, offset, numVoxel))
	{
		VoxelSegmentationInfo & v = pVoxelInfo[offset];
		v.m_numMarkedCameras = 0;

		for(uint camIndex = 0; camIndex < v.m_numImages; ++camIndex)
		{
			const TemporaryDeviceVector<uint2> & imgPointsSingleCam = v.getImgPoints(camIndex);
			const uint * pCurrentImage = pImages + camIndex * (sizeImages.x * sizeImages.y);

			uint segmentationCounter = 0;

			for(size_t i = 0; i < imgPointsSingleCam.size(); i += 2)
			{
				const uint * pStart = pCurrentImage + imgPointsSingleCam[i].y     * sizeImages.x + imgPointsSingleCam[i].x; 
				const uint * pEnd   = pCurrentImage + imgPointsSingleCam[i + 1].y * sizeImages.x + imgPointsSingleCam[i + 1].x; 

				segmentationCounter += *pEnd - *pStart;
			}
			const uint segmentationThreshold = v.getNumImgPoints(camIndex) * minSegmentation;

			const VoxelSegmentationStatus s = segmentationCounter > segmentationThreshold ? Marked : NotMarked;
			v.m_segmentationStats[camIndex] = s;
			if(s == Marked)
			{
				++v.m_numMarkedCameras;
			}
		}

		const bool active = voxelPredicate_maximumVisibleActive(v);
		pVoxel[offset].m_isActive = active;
	}
}


__global__ void updateSpaceFastFromIntegralImage(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxel, uint * pImages, uint2 sizeImages, uint numImages, float minSegmentation)
{
	const uint voxelId_x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint voxelId_y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint voxelId_z = blockIdx.z;
	const uint camIndex  = threadIdx.z;
	
	const uint offset = voxelId_x * numVoxel.y * numVoxel.z + voxelId_y * numVoxel.z + voxelId_z; 

	if(isValidVoxelIdx(voxelId_x, voxelId_y, voxelId_z, offset, numVoxel))
	{
		VoxelSegmentationInfo & v = pVoxelInfo[offset];

		const TemporaryDeviceVector<uint2> & imgPointsSingleCam = v.getImgPoints(camIndex);
		const uint * pCurrentImage = pImages + camIndex * (sizeImages.x * sizeImages.y);

		uint segmentationCounter = 0;

		for(size_t i = 0; i < imgPointsSingleCam.size(); i += 2)
		{
			const uint * pStart = pCurrentImage + imgPointsSingleCam[i].y     * sizeImages.x + imgPointsSingleCam[i].x; 
			const uint * pEnd   = pCurrentImage + imgPointsSingleCam[i + 1].y * sizeImages.x + imgPointsSingleCam[i + 1].x; 

			segmentationCounter += *pEnd - *pStart;
		}
		const uint segmentationThreshold = v.getNumImgPoints(camIndex) * minSegmentation;

		const VoxelSegmentationStatus s = segmentationCounter > segmentationThreshold ? Marked : NotMarked;
		v.m_segmentationStats[camIndex] = s;
	}

	__syncthreads();

	if(isValidVoxelIdx(voxelId_x, voxelId_y, voxelId_z, offset, numVoxel) && camIndex == 0)
	{		
		VoxelSegmentationInfo & v = pVoxelInfo[offset];
		const bool active = voxelPredicate_maximumVisibleActive(v);
		pVoxel[offset].m_isActive = active;
	}
}


__global__ void updateSpaceFromSurfaceIntegralImage(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxel, hipSurfaceObject_t images, uint2 sizeImages, float minSegmentation)
{
	const uint voxelId_x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint voxelId_y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint voxelId_z = blockIdx.z * blockDim.z + threadIdx.z;
	
	const uint offset = voxelId_x * numVoxel.y * numVoxel.z + voxelId_y * numVoxel.z + voxelId_z; 

	if(isValidVoxelIdx(voxelId_x, voxelId_y, voxelId_z, offset, numVoxel))
	{
		VoxelSegmentationInfo & v = pVoxelInfo[offset];
		v.m_numMarkedCameras = 0;

		for(uint camIndex = 0; camIndex < v.m_numImages; ++camIndex)
		{
			const TemporaryDeviceVector<uint2> & imgPointsSingleCam = v.getImgPoints(camIndex);
			const uint offsetCurrentImage = camIndex * sizeImages.x;

			uint segmentationCounter = 0;

			for(size_t i = 0; i < imgPointsSingleCam.size(); i += 2)
			{
				uint start, end;
				surf2Dread(&start, images, (offsetCurrentImage + imgPointsSingleCam[i].x) * sizeof(uint), imgPointsSingleCam[i].y ); 
				surf2Dread(&end, images, (offsetCurrentImage + imgPointsSingleCam[i+1].x) * sizeof(uint), imgPointsSingleCam[i+1].y ); 

				segmentationCounter += end - start;
			}
			const uint segmentationThreshold = v.getNumImgPoints(camIndex) * minSegmentation;

			VoxelSegmentationStatus s = segmentationCounter > segmentationThreshold ? Marked : NotMarked;
			v.m_segmentationStats[camIndex] = s;
			if(s == Marked)
			{
				++v.m_numMarkedCameras;
			}
		}

		const bool active = voxelPredicate_maximumVisibleActive(v);
		pVoxel[offset].m_isActive = active;
	}
}


__global__ void updateSpaceFastFromSurfaceIntegralImage(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxel, hipSurfaceObject_t images, uint2 sizeImages, uint numImages, float minSegmentation)
{
	const uint voxelId_x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint voxelId_y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint voxelId_z = blockIdx.z;
	const uint camIndex  = threadIdx.z;
	
	const uint offset = voxelId_x * numVoxel.y * numVoxel.z + voxelId_y * numVoxel.z + voxelId_z; 

	if(isValidVoxelIdx(voxelId_x, voxelId_y, voxelId_z, offset, numVoxel))
	{
		VoxelSegmentationInfo & v = pVoxelInfo[offset];

		const TemporaryDeviceVector<uint2> & imgPointsSingleCam = v.getImgPoints(camIndex);
		const uint offsetCurrentImage = camIndex * sizeImages.x;

		uint segmentationCounter = 0;

		for(size_t i = 0; i < imgPointsSingleCam.size(); i += 2)
		{
			uint start, end;
			surf2Dread(&start, images, (offsetCurrentImage + imgPointsSingleCam[i].x) * sizeof(uint), imgPointsSingleCam[i].y ); 
			surf2Dread(&end, images, (offsetCurrentImage + imgPointsSingleCam[i+1].x) * sizeof(uint), imgPointsSingleCam[i+1].y ); 

			segmentationCounter += end - start;
		}
		const uint segmentationThreshold = v.getNumImgPoints(camIndex) * minSegmentation;

		VoxelSegmentationStatus s = segmentationCounter > segmentationThreshold ? Marked : NotMarked;
		v.m_segmentationStats[camIndex] = s;
	}

	__syncthreads();

	if(isValidVoxelIdx(voxelId_x, voxelId_y, voxelId_z, offset, numVoxel) && camIndex == 0)
	{		
		VoxelSegmentationInfo & v = pVoxelInfo[offset];
		const bool active = voxelPredicate_maximumVisibleActive(v);
		pVoxel[offset].m_isActive = active;
	}
}


__global__ void updateSpaceFastFromSurfaceIntegralImage_updateSegmentation(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxel, hipSurfaceObject_t images, uint2 sizeImages, uint numImages, float minSegmentation)
{
	const uint voxelId_x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint voxelId_y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint voxelId_z = blockIdx.z / numImages;
	const uint camIndex  = blockIdx.z % numImages;

	const uint offset = voxelId_x * numVoxel.y * numVoxel.z + voxelId_y * numVoxel.z + voxelId_z; 

	if(isValidVoxelIdx(voxelId_x, voxelId_y, voxelId_z, offset, numVoxel))
	{
		VoxelSegmentationInfo & v = pVoxelInfo[offset];

		const TemporaryDeviceVector<uint2> & imgPointsSingleCam = v.getImgPoints(camIndex);
		const uint offsetCurrentImage = camIndex * sizeImages.x;

		uint segmentationCounter = 0;

		for(size_t i = 0; i < imgPointsSingleCam.size(); i += 2)
		{
			uint start, end;
			surf2Dread(&start, images, (offsetCurrentImage + imgPointsSingleCam[i].x) * sizeof(uint), imgPointsSingleCam[i].y ); 
			surf2Dread(&end, images, (offsetCurrentImage + imgPointsSingleCam[i+1].x) * sizeof(uint), imgPointsSingleCam[i+1].y ); 

			segmentationCounter += end - start;
		}
		const uint segmentationThreshold = v.getNumImgPoints(camIndex) * minSegmentation;

		VoxelSegmentationStatus s = segmentationCounter > segmentationThreshold ? Marked : NotMarked;
		v.m_segmentationStats[camIndex] = s;
	}
}



__global__ void updateVoxelOnly(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxel)
{
	const uint voxelId_x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint voxelId_y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint voxelId_z = blockIdx.z;

	const uint offset = voxelId_x * numVoxel.y * numVoxel.z + voxelId_y * numVoxel.z + voxelId_z; 

	if(isValidVoxelIdx(voxelId_x, voxelId_y, voxelId_z, offset, numVoxel))
	{		
		VoxelSegmentationInfo & v = pVoxelInfo[offset];
		const bool active = voxelPredicate_maximumVisibleActive(v);
		pVoxel[offset].m_isActive = active;
	}
}


void call_updateSpace(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxels, unsigned char * pImages, uint2 sizeImages, float minSegmentation)
{

	dim3 blockSize(32, 32, 1);
	dim3 gridSize((numVoxels.x + 31) / 32, (numVoxels.y + 31) / 32, numVoxels.z);

	updateSpace<<<gridSize, blockSize>>>(pVoxelInfo, pVoxel, numVoxels, pImages, sizeImages, minSegmentation);
	cudaCheckError();
}


void call_updateSpaceFast(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxels, unsigned char * pImages, uint2 sizeImages, uint numImages, float minSegmentation)
{
	assert(numImages == 4);
	dim3 blockSize(16, 16, numImages);
	dim3 gridSize((numVoxels.x + 15) / 16, (numVoxels.y + 15) / 16, numVoxels.z);

	updateSpaceFast<<<gridSize, blockSize>>>(pVoxelInfo, pVoxel, numVoxels, pImages, sizeImages, numImages, minSegmentation);
	cudaCheckError();
}


void call_updateSpaceFromIntegralImage(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxels, uint * pImages, uint2 sizeImages, float minSegmentation)
{
	dim3 blockSize(32, 32, 1);
	dim3 gridSize((numVoxels.x + 31) / 32, (numVoxels.y + 31) / 32, numVoxels.z);

	updateSpaceFromIntegralImage<<<gridSize, blockSize>>>(pVoxelInfo, pVoxel, numVoxels, pImages, sizeImages, minSegmentation);
	cudaCheckError();
}


void call_updateSpaceFastFromIntegralImage(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxels, uint * pImages, uint2 sizeImages, uint numImages, float minSegmentation)
{
	assert(numImages == 4);
	dim3 blockSize(16, 16, numImages);
	dim3 gridSize((numVoxels.x + 15) / 16, (numVoxels.y + 15) / 16, numVoxels.z);

	updateSpaceFastFromIntegralImage<<<gridSize, blockSize>>>(pVoxelInfo, pVoxel, numVoxels, pImages, sizeImages, numImages, minSegmentation);
	cudaCheckError();
}


void call_updateSpaceFromSurfaceIntegralImage(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxels, hipSurfaceObject_t images, uint2 sizeImages, float minSegmentation)
{
	dim3 blockSize(32, 32, 1);
	dim3 gridSize((numVoxels.x + 31) / 32, (numVoxels.y + 31) / 32, numVoxels.z);

	updateSpaceFromSurfaceIntegralImage<<<gridSize, blockSize>>>(pVoxelInfo, pVoxel, numVoxels, images, sizeImages, minSegmentation);
	cudaCheckError();
}


void call_updateSpaceFastFromSurfaceIntegralImage(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxels, hipSurfaceObject_t images, uint2 sizeImages, uint numImages, float minSegmentation)
{
	assert(numImages == 4);
	dim3 blockSize(16, 16, numImages);
	dim3 gridSize((numVoxels.x + 15) / 16, (numVoxels.y + 15) / 16, numVoxels.z);

	updateSpaceFastFromSurfaceIntegralImage<<<gridSize, blockSize>>>(pVoxelInfo, pVoxel, numVoxels, images, sizeImages, numImages, minSegmentation);
	cudaCheckError();
}


void call_updateSpaceFastFromSurfaceIntegralImage_2parts(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxels, hipSurfaceObject_t images, uint2 sizeImages, uint numImages, float minSegmentation)
{
	dim3 blockSize(32, 32, 1);
	dim3 gridSize((numVoxels.x + 31) / 32, (numVoxels.y + 31) / 32, numVoxels.z * numImages);

	updateSpaceFastFromSurfaceIntegralImage_updateSegmentation<<<gridSize, blockSize>>>(pVoxelInfo, pVoxel, numVoxels, images, sizeImages, numImages, minSegmentation);
	cudaCheckError();

	dim3 gridSizeUpdateVoxel((numVoxels.x + 31) / 32, (numVoxels.y + 31) / 32, numVoxels.z);

	updateVoxelOnly<<<gridSizeUpdateVoxel, blockSize>>>(pVoxelInfo, pVoxel, numVoxels);
	cudaCheckError();
}


void call_updateSpace_updateVoxel(VoxelSegmentationInfo * pVoxelInfo, Voxel * pVoxel, uint3 numVoxels)
{
	dim3 blockSize(32, 32, 1);
	dim3 gridSizeUpdateVoxel((numVoxels.x + 31) / 32, (numVoxels.y + 31) / 32, numVoxels.z);

	updateVoxelOnly<<<gridSizeUpdateVoxel, blockSize>>>(pVoxelInfo, pVoxel, numVoxels);
	cudaCheckError();
}
